#include "hip/hip_runtime.h"
#include <torch/extension.h>

typedef torch::PackedTensorAccessor32<int32_t, 2, torch::RestrictPtrTraits> int_2d;
typedef torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits> float_2d;
typedef torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> float_3d;


__global__ void weighted_sum_kernel(
    const float_3d x,
    const int_2d group,
    const float_2d weights,
    float_3d y
) {
    int B = x.size(0);
    int N = x.size(1);
    int D = x.size(2);
    int C = y.size(1);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int b_idx = idx / N;
    int n_idx = idx % N;
    if (b_idx >= B) return;

    int c_idx = group[b_idx][n_idx];
    if (c_idx < 0 || c_idx >= C) return;

    float w = weights[b_idx][c_idx];
    for (int d_idx = 0; d_idx < D; d_idx++) {
        atomicAdd(&y[b_idx][c_idx][d_idx], x[b_idx][n_idx][d_idx] * w);
    }
}

void weighted_sum(
    const torch::Tensor x,
    const torch::Tensor group,
    const torch::Tensor weights,
    torch::Tensor y
) {
    int B = x.size(0);
    int N = x.size(1);
    int D = x.size(2);
    int C = y.size(1);

    const int threads = 1024;
    int blocks = (B*N - 1) / threads + 1;

    weighted_sum_kernel<<<blocks, threads>>>(
        x.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        group.packed_accessor32<int32_t, 2, torch::RestrictPtrTraits>(),
        weights.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
        y.packed_accessor32<float, 3, torch::RestrictPtrTraits>()
    );
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("_weighted_sum", &weighted_sum);
}
